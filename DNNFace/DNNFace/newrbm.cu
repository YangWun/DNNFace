#include "hip/hip_runtime.h"
#include "./inc/connection.h" 
#include "./inc/rbm.h" 
#include "./inc/trainer.h" 
#include <hiprand.h>
#include "cuPrintf.cu"

using namespace utilLearn; 


//RBM LEARNING PARAMETERS
//>--SAMPLESΪGIBS_SAMPLES
#define BATCH      100 
#define SAMPLES      1 
#define STEPS      1 
#define EPOCH      200 

//RBM SIZE PARAMETERS 
#define  VSIZE_X      96 
#define VSIZE_Y      96 
#define VSIZE      9216 


//#define  HSIZE_X      64 
//#define HSIZE_Y      64 
//#define HSIZE      4096 
#define  HSIZE_X      32 
#define HSIZE_Y      32 
#define HSIZE      1024 



char * const param_file = "params/norb-persistent-lvl1.rbm"; 
char * const converted_data_file = "data/norb-images-lvl1.floats";

char * const data_file = "data/smallnorb-5x46789x9x18x6x2x96x96-training-dat.mat"; 

//For Transpose 
#define TILE_DIM   16 
#define BLOCK_ROWS    16 
#define  BLOCKS_LAYER  16 


void train_novis(); 


//Helper Functions 
void init_fantasy(); 
void train_mini_persistent(); 
void update_params(); 
void convert(char* out_file); 

//CUDA Functions 
__global__ void transpose(float *w, float *wt); 
__global__ void upPassInitProb(float* v0, float* h0, float* b, float* w); 
__global__ void upPassInit(float* v0, float* h0, float* b, float* w, float* rnd); 
__global__ void upPass(float* vX, float* hX, float* b, float* w, float* rnd); 
__global__ void upPassProb(float* vX, float* hX, float* b, float* w); 
__global__ void downPass(float* vX, float* hX, float* a, float* wt); 
__global__ void updateW(float* v0, float* vX, float* h0, float* hX, float* w, float* vel_w, float momentum, float* q, float* dw, float l_rate); 
__global__ void updateA(float* v0, float* vX, float* a, float l_rate); 
__global__ void updateB(float* h0, float* hX, float* b, float* q, float l_rate); 


//Globals 
Rbm *my_rbm; 
Trainer *my_trainer;
hiprandGenerator_t d_rand; 

//float total_time; 


int main(int argc, char** argv) 
{ 

	//total_time = 0; 
	//Set GPU 1 (currently not used for display) 
	hipSetDevice(1); 

	//Set up basic units 
	Connection* my_connection = new Connection(VSIZE, HSIZE); 
	Layer* my_visible = new Layer(VSIZE_X, VSIZE_Y, BATCH, SAMPLES, false); 
	Layer* my_hidden = new Layer(HSIZE_X, HSIZE_Y, BATCH, SAMPLES, true); 
 
	my_rbm = new Rbm(my_visible, my_hidden, my_connection); 
	my_trainer = new Trainer(BATCH, SAMPLES, EPOCH, 0.001, 10, 0.5, 0.0001); 
	//Load Data Set 
	if(my_trainer->loadTrainingDataMAT(data_file) < 0) 
	{ 
		printf("An error occurred loading the training data. Exiting...\n"); 
		return -1; 
	}


	//Set up RBM Parameters 

	printf("Initializing Paramters\n"); 
	my_rbm->initParams(); 
	dim3 grid(my_rbm->getHSize()/TILE_DIM, my_rbm->getVSize()/TILE_DIM), threads(TILE_DIM,BLOCK_ROWS); 
	transpose<<<grid,threads>>>(my_rbm->getDevW(), my_rbm->getDevWT()); 

	printf("Setting visual bias to data probability\n"); 
	//Set visual bias to training data probability 
	for(int i=0;i<my_connection->getVSize();i++) 
	{ 
		//printf("visual bias[%d]=%f\n",i, my_trainer->pixelProb(i)); 
		my_connection->setA(i,my_trainer->pixelProb(i)); 
	} 
	my_connection->cpyA(); //Place on device 

	//Set up Random Initializer 
	hiprandCreateGenerator(&d_rand, HIPRAND_RNG_PSEUDO_MTGP32); 
	srand((unsigned)time(0)); 
	int seed = (rand() % 1000); 
	hiprandSetPseudoRandomGeneratorSeed(d_rand, seed); 

	init_fantasy(); 

	while(!my_trainer->trainComplete()) 
	{ 
		train_mini_persistent(); 
 
		//Update training status 
		my_trainer->incN(); 
		if(my_trainer->epochComplete()) 
		{ 
			printf("Epoch %d Complete!\n", my_trainer->getEpoch()); 
			//my_rbm->save(param_file); 
			//saveFreeNRG(); 
			my_rbm->checkSparsityH(); 
		} 

	} 

	printf("Training run complete!\n"); 
	my_rbm->save(param_file);

	//train_novis(); 
	return 0; 
} 

//>--Used for Persistent CD
void init_fantasy() 
{ 
	my_trainer->randBatchV(); 
	hiprandGenerateUniform(d_rand, (float *) my_rbm->getHrand(), my_rbm->getHSize()*my_trainer->getNumFantasy()); 

	//Calculate HX 
	dim3 blockDim(BLOCKS_LAYER,SAMPLES,BATCH); 
	dim3 threadDimUp(my_rbm->getHSize()/BLOCKS_LAYER);
	//>--���������ݳ�ʼ��HX
	upPassInit<<<blockDim,threadDimUp>>>(my_trainer->d_mini_batch_data,my_rbm->getHX(),my_rbm->getDevB(),my_rbm->getDevW(), my_rbm->getHrand()); 
} 


void train_novis() 
{ 


} 


void train_mini_persistent() 
{ 
	//Select Batch Samples V0 
	my_trainer->randBatchV(); 
	hiprandGenerateUniform(d_rand, (float *) my_rbm->getHrand(), my_rbm->getHSize()*my_trainer->getNumFantasy()); 



	//Calculate H0 (��һС�δ������ע�͵���--->����ע�͵�����Ϊ�ڸ���Ȩ��ʱ��ҪH0)
	dim3 blockDim(BLOCKS_LAYER,SAMPLES,BATCH); 
	dim3 threadDimUp(my_rbm->getHSize()/BLOCKS_LAYER); 
	dim3 threadDimDown(my_rbm->getVSize()/BLOCKS_LAYER); 
	upPassInit<<<blockDim,threadDimUp>>>(my_trainer->d_mini_batch_data,my_rbm->getH0(),my_rbm->getDevB(),my_rbm->getDevW(), my_rbm->getHrand()); 

	//Calculate V1
	//>--���ϴε�������ģ�Ͳ���HX����V1(Persistent CD �㷨)
	downPass<<<blockDim,threadDimDown>>>(my_rbm->getVX(),my_rbm->getHX(),my_rbm->getDevA(),my_rbm->getDevWT()); 


	//Iterate over gibbs steps HX and VX 
	//for (int g=1;g<STEPS;g++) 
	//{ 
	//	hiprandGenerateUniform(d_rand, (float *) my_rbm->getHrand(),my_rbm->getHSize()*my_trainer->getNumFantasy()); 
	//	upPass<<<blockDim,threadDimUp>>>(my_rbm->getVX(),my_rbm->getHX(),my_rbm->getDevB(),my_rbm->getDevW(), my_rbm->getHrand()); 
	//	downPass<<<blockDim,threadDimDown>>>(my_rbm->getVX(),my_rbm->getHX(),my_rbm->getDevA(),my_rbm->getDevWT()); 
	//} 



	//Calculate HX (probabilities for update)
	//>--��V1����HX (��ʱ��HX�Ǹ���ֵ������W, a, b�ĸ��£���0/1��ֵ)
	upPassProb<<<blockDim,threadDimUp>>>(my_rbm->getVX(),my_rbm->getHX(),my_rbm->getDevB(),my_rbm->getDevW()); 

	float *tmp4=(float*) malloc(my_rbm->getHSize() *my_trainer->getNumFantasy()* sizeof(float));
	hipMemcpy(tmp4, my_rbm->getHX(), my_rbm->getHSize() * my_trainer->getNumFantasy()*sizeof(float), hipMemcpyDeviceToHost);
	for(int i=0;i< my_rbm->getHSize();i++){
		cout<<tmp4[i]<<endl;
	}

	update_params(); 


	//>--���㵱ǰ������ģ�Ͳ���HX(0/1��ֵ)���´ε���ʹ��(Persistent CD �㷨)
	upPass<<<blockDim,threadDimUp>>>(my_rbm->getVX(),my_rbm->getHX(),my_rbm->getDevB(),my_rbm->getDevW(), my_rbm->getHrand()); 

	bool *tmp=(bool*) malloc(my_rbm->getHSize() * sizeof(bool));
	hipMemcpy(tmp, my_rbm->getH0(), my_rbm->getHSize() * sizeof(bool), hipMemcpyDeviceToHost);
	for(int i=0;i< my_rbm->getHSize();i++){
		cout<<tmp[i]<<endl;
	}

	return; 
} 

void update_params() 
{ 


	//Update Parameters 
	dim3 threadDimUp(my_rbm->getHSize()/BLOCKS_LAYER); 
	dim3 threadDimDown(my_rbm->getVSize()/BLOCKS_LAYER); 
	dim3 updateBlockDim(BLOCKS_LAYER,my_rbm->getVSize());

	//my_rbm->checkSparsityH();
	updateW<<<updateBlockDim,threadDimUp>>>(my_trainer->d_mini_batch_data,my_rbm->getVX(),my_rbm->getH0(),my_rbm->getHX(),my_rbm->getDevW() 
		,my_rbm->getDevVw(), my_trainer->getMomentum(), my_rbm->getHQ(), my_rbm->getDevDw(), my_trainer->getLearnRate()); 
	//my_rbm->checkSparsityH();

	float *tmp3=(float*) malloc(my_rbm->getHSize() * my_rbm->getVSize() * sizeof(float));
	hipMemcpy(tmp3, my_rbm->getDevW(), my_rbm->getHSize() * my_rbm->getVSize() * sizeof(float), hipMemcpyDeviceToHost);
	for(int i=0;i< my_rbm->getHSize() * my_rbm->getVSize();i++){
		cout<<tmp3[i]<<endl;
	}

	dim3 grid(my_rbm->getHSize()/TILE_DIM, my_rbm->getVSize()/TILE_DIM);
	dim3 threads(TILE_DIM,BLOCK_ROWS); 
	transpose<<<grid,threads>>>(my_rbm->getDevW(), my_rbm->getDevWT());

	float *tmp4=(float*) malloc(my_rbm->getHSize() * my_rbm->getVSize() * sizeof(float));
	hipMemcpy(tmp4, my_rbm->getDevWT(), my_rbm->getHSize() * my_rbm->getVSize() * sizeof(float), hipMemcpyDeviceToHost);
	//for(int i=0;i< my_rbm->getHSize() * my_rbm->getVSize();i++){
	//	cout<<tmp4[i]<<endl;
	//}
	for(int i=0;i<my_rbm->getVSize();i++)
		for(int j=0;j<my_rbm->getHSize();j++){
			cout<<tmp3[i*my_rbm->getHSize()+j]<<" "<<tmp4[j*my_rbm->getVSize()+i]<<endl;
		}


	//float *tmp4=(float*) malloc(my_rbm->getHSize() * sizeof(float));
	//hipMemcpy(tmp4, my_rbm->getHrand(), my_rbm->getHSize() * sizeof(float), hipMemcpyDeviceToHost);
	//for(int i=0;i< my_rbm->getHSize();i++){
	//	cout<<tmp4[i]<<endl;
	//}

	updateA<<<BLOCKS_LAYER,threadDimDown>>>(my_trainer->d_mini_batch_data,my_rbm->getVX(),my_rbm->getDevA(),my_trainer->getLearnRate()); 
	updateB<<<BLOCKS_LAYER,threadDimUp>>>(my_rbm->getH0(),my_rbm->getHX(),my_rbm->getDevB(), my_rbm->getHQ(), my_trainer->getLearnRate()); 
	return;
} 

void convert(char* out_file) 
{ 

	printf("Converting Data...\n"); 
	ofstream o_file; 
	o_file.open(out_file, ios::binary); 

	int loc = 0; 
	if(o_file.is_open()) 
	{ 
		//Save number of training images 
		int num = my_trainer->getTrainSize(); 
		o_file.seekp(loc); 
		o_file.write((char*)&num, sizeof(int)); 

		loc += sizeof(int); 

		loc = my_rbm->saveHDim(&o_file, loc); 

		dim3 blockDim(BLOCKS_LAYER,1,1); 
		dim3 threadDimUp(my_rbm->getHSize()/BLOCKS_LAYER); 

		for(int i=0;i<num;i++) 
		{ 
			//printf("Converting Image: %d\n",i); 
			//Select Batch Samples V0 
			my_trainer->setV(i,0); 

			//Calculate H0 


			upPassInitProb<<<blockDim,threadDimUp>>>(my_trainer->d_mini_batch_data,my_rbm->getHX(),my_rbm->getDevB(),my_rbm->getDevW()); 
			loc = my_rbm->saveH(&o_file,loc); 


		} 
		o_file.close(); 
		printf("Completed\n"); 
	} 
	else 
		printf("Failed\n"); 
	return; 

} 

/*==================================================================
= 
*       CUDA FUNCTIONS 
===================================================================*
/ 

/* --------------------------------------------------------------- 
*    UP PASS INIT 
* Initial V0->H0 pass. This is necessarily different because 
* all fantasy particles use the same initial V0. 
* 
* v0 | float* | Training examples 
* h0 | float* | Hidden Layers to calculate 
* b  | float* | Bias to hidden units 
* w  | float* | Weights 
* rnd| float* | Random vectors to compete H prob to 
------------------------------------------------------------------*/ 
__global__ void upPassInit(float* v0, float* h0, float* b, float* w, 
	float* rnd) 
{ 
	int h_idx = (blockIdx.x * blockDim.x) + threadIdx.x; 
	int g_off = blockIdx.y; 
	int b_off = blockIdx.z; 
	int t_off = ( (b_off * gridDim.y + g_off) * HSIZE ) + h_idx; 

	float sum = b[h_idx]; 
	//printf("sum = %f \n",b[h_idx]); 
	for(int i=0;i<VSIZE;i++) 
	{ 
		sum += v0[b_off*VSIZE + i] * w[ i*HSIZE + h_idx]; 
	} 
	//printf("sum = %f \n",b[h_idx]); 
	float prob = 1 / (1 + __expf(-1 * sum)); 

	//printf("p(H[%d]=1|v) = %f > %f\n",h_idx, prob, rnd[h_idx + b_offset]); 
	h0[t_off] = (prob > rnd[t_off]); 
} 

__global__ void upPassInitProb(float* v0, float* h0, float* b, 
	float* w) 
{ 
	int h_idx = (blockIdx.x * blockDim.x) + threadIdx.x; 
	int g_off = blockIdx.y; 
	int b_off = blockIdx.z; 
	int t_off = ( (b_off * gridDim.y + g_off) * HSIZE ) + h_idx; 

	float sum = b[h_idx]; 
	//printf("sum = %f \n",b[h_idx]); 
	for(int i=0;i<VSIZE;i++) 
	{ 
		sum += v0[b_off*VSIZE + i] * w[ i*HSIZE + h_idx]; 
	} 
	//printf("sum = %f \n",b[h_idx]); 
	h0[t_off] = 1 / (1 + __expf(-1 * sum)); 
} 

/* --------------------------------------------------------------- 
*    UP PASS 
* Any VX->HX pass. Output is Binary. 
* 
* vX | float* | Visible Layers to use 
* hX | float* | Hidden Layers to calculate 
* b  | float* | Bias to hidden units 
* w  | float* | Weights 
* rnd| float* | Random vectors to compete H prob to 
------------------------------------------------------------------*/ 
__global__ void upPass(float* vX, float* hX, float* b, float* w, float* rnd) 
{ 
	int h_idx = (blockIdx.x * blockDim.x) + threadIdx.x; 
	int g_off = blockIdx.y; 
	int b_off = blockIdx.z * gridDim.y; 
	int t_off = ( (b_off + g_off) * HSIZE ) + h_idx; 

	float sum = b[h_idx]; 
	//printf("sum = %f \n",b[h_idx]); 
	for(int i=0;i<VSIZE;i++) 
	{ 
		sum += vX[(b_off + g_off)*VSIZE + i] * w[ i*HSIZE + 
			h_idx]; 
	} 
	//printf("sum = %f \n",b[h_idx]); 
	float prob = 1 / (1 + __expf(-1 * sum)); 

	//printf("p(H[%d]=1|v) = %f > %f\n",h_idx, prob, rnd[h_idx + b_offset]); 
	hX[t_off] = (prob > rnd[t_off]); 
} 

/* --------------------------------------------------------------- 
*    UP PASS PROB 
* Final VX->HX pass. Output is probability. 
* 
* vX | float* | Visible Layers to use 
* hX | float* | Hidden Layers to calculate 
* b  | float* | Bias to hidden units 
* w  | float* | Weights 
------------------------------------------------------------------*/ 
__global__ void upPassProb(float* vX, float* hX, float* b, float* w) 
{ 
	int h_idx = (blockIdx.x * blockDim.x) + threadIdx.x; 
	int g_off = blockIdx.y;
	int b_off = blockIdx.z * gridDim.y; 
	int t_off = ( (b_off + g_off) * HSIZE ) + h_idx; 

	float sum = b[h_idx]; 
	//printf("sum = %f \n",b[h_idx]); 
	for(int i=0;i<VSIZE;i++) 
	{ 
		sum += vX[(b_off + g_off)*VSIZE + i] * w[ i*HSIZE + 
			h_idx]; 
	} 
	//printf("sum = %f \n",b[h_idx]); 
	hX[t_off] = 1 / (1 + __expf(-1 * sum)); 
} 

/* --------------------------------------------------------------- 
*    DOWN PASS 
* Any HX->VX pass. Output is probability. 
* 
* vX | float* | Visible Layers to calculate 
* hX | float* | Hidden Layers to use 
* a  | float* | Bias to visible units 
* wt | float* | Weights Transposed 
------------------------------------------------------------------*/ 
__global__ void downPass(float* vX, float* hX, float* a, float* wt) 
{ 
	int v_idx = (blockIdx.x * blockDim.x) + threadIdx.x; 
	int g_off = blockIdx.y; 
	int b_off = blockIdx.z * gridDim.y; 
	int t_off = ( (b_off + g_off) * VSIZE ) + v_idx; 

	float sum = a[v_idx]; 
	//printf("sum = %f \n",b[h_idx]); 
	for(int i=0;i<HSIZE;i++) 
	{ 
		//sum += hX[b_off + g_off + i] * w[ i*512 + v_idx]; 
		sum += hX[(b_off + g_off)*HSIZE + i] * wt[ i*VSIZE + v_idx]; 
	} 
	//printf("sum = %f \n",b[h_idx]); 
	vX[t_off] = 1 / (1 + __expf(-1 * sum)); 

} 

/* --------------------------------------------------------------- 
*    UPDATE W 
* Calculates the change to the weights 
* 
* v0     | float* | Visible layer from data 
* vX     | float* | Final Visible layer from model 
* h0     | float* | Hidden layer one pass from data 
* hX     | float* | Hidden layer from model 
* w      | float* | Weights
* momentum | float* | ����ѧϰ��
* q       | float* | problity
* dw      | float* | weights updates
* l_rate  | float  | learning rate 
------------------------------------------------------------------*/ 
__global__ void updateW(float* v0, float* vX, float* h0, float* hX, 
	float* w, float* vel_w, float momentum, float* q, float* dw, float 
	l_rate) 
{ 
	int h_idx = (blockIdx.x * blockDim.x) + threadIdx.x; 
	int v_idx = (blockIdx.y); 
	int v_offset = v_idx * blockDim.x * gridDim.x; 

	float delta = 0.0; 
	float sum_h = 0.0; 

	for(int batch=0;batch<BATCH;batch++) 
	{ 
		for(int gibbs=0;gibbs<SAMPLES;gibbs++) 
		{ 
			int h_off = h_idx + batch*SAMPLES*HSIZE + 
				gibbs*HSIZE; 
			int v_off = v_idx + batch*SAMPLES*VSIZE + 
				gibbs*VSIZE; 

			delta += (v0[v_idx + batch*VSIZE] * h0[h_off]) - 
				(vX[v_off] * hX[h_off]); 
			sum_h += (hX[h_off]); 
		} 
	} 

	//Calculate probability estimate 
	//>--(sum_h / (BATCH * SAMPLES)) is the current estimated sparsity problity
	q[h_idx] = ((.95)*q[h_idx]) + (1-.95)*(sum_h / (BATCH * SAMPLES)); 



	//if(v_idx == 200 && h_idx < 5) 
	//printf("Q = %f\n", q[h_idx]); 
	//if(h_idx + v_offset == 555) 
	//printf("w[%d]=%f += %f\n", h_idx + v_offset, w[h_idx + v_offset], delta); 

	//VELOCITY 
	vel_w[h_idx + v_offset] = momentum * vel_w[h_idx + v_offset] + ( (delta * l_rate) / (SAMPLES * BATCH) ); 

	//DECAY 
	float decay = (0.0005 * w[h_idx + v_offset] ) * l_rate; 


	//SPARSITY 
	// = penalty * ( probability estimation - probability target) 
	float sparsity = 0.0001 * (q[h_idx]-0.1); 

	dw[h_idx + v_offset] = (vel_w[h_idx + v_offset] - decay - sparsity); 
	w[h_idx + v_offset] += (vel_w[h_idx + v_offset] - decay - sparsity); 

	//VELOCITY AND SPARSITY ONLY 
	//dw[h_idx + v_offset] = (vel_w[h_idx + v_offset]  - sparsity); 
	//w[h_idx + v_offset] += (vel_w[h_idx + v_offset]  - sparsity); 

	//w[h_idx + v_offset] += (delta * l_rate) / (SAMPLES * BATCH); 
	//w[h_idx + v_offset] = delta; 

	//dw[h_idx + v_offset] = ((delta * l_rate) / (SAMPLES * BATCH) ) - (decay * l_rate); 
	//w[h_idx + v_offset] += ((delta * l_rate) / (SAMPLES * BATCH) ) - (decay * l_rate); 
} 

/* --------------------------------------------------------------- 
*    UPDATE A 
* Calculates the change to the visible bias 
* 
* v0     | float* | Visible layer from data 
* vX     | float* | Final Visible layer from model 
* a      | float* | Visible bias 
* l_rate  | float  | learning rate 
------------------------------------------------------------------*/ 
__global__ void updateA(float* v0, float* vX, float* a, float 
	l_rate) 
{ 
	int v_idx = (blockIdx.x * blockDim.x) + threadIdx.x; 

	float delta = 0.0; 

	for(int batch=0;batch<BATCH;batch++) 
	{ 
		for(int gibbs=0;gibbs<SAMPLES;gibbs++) 
		{ 
			int v_off = v_idx + batch*SAMPLES*VSIZE + 
				gibbs*VSIZE; 
			delta += (v0[v_idx + batch*VSIZE]) - (vX[v_off]); 
		} 
	} 

	a[v_idx] += ( (delta * l_rate) / (SAMPLES * BATCH) ); 
}
/* --------------------------------------------------------------- 
*    UPDATE B 
* Calculates the change to the hidden bias 
* 
* h0     | float* | Hidden layer one pass from data 
* hX     | float* | Hidden layer from model 
* b      | float* | Hidden bias 
* l_rate  | float  | learning rate 
------------------------------------------------------------------*/ 
__global__ void updateB(float* h0, float* hX, float* b, float* q, 
	float l_rate) 
{ 
	int h_idx = (blockIdx.x * blockDim.x) + threadIdx.x; 

	float delta = 0.0; 

	for(int batch=0;batch<BATCH;batch++) 
	{ 
		for(int gibbs=0;gibbs<SAMPLES;gibbs++) 
		{ 
			int h_off = h_idx + batch*SAMPLES*HSIZE + 
				gibbs*HSIZE; 
			delta += (h0[h_off]) - (hX[h_off]); 
		} 
	} 

	//float sparsity = (0.0001 * ((q[h_idx]-0.04)*(q[h_idx]-0.04)) ); 
	// = penalty * ( probability estimation - probability target) 
	float sparsity = 0.0001 * (q[h_idx]-0.1); 

	//if(h_idx < 5) 
	//printf("sparsity penalty = %f\n",sparsity); 

	b[h_idx] += ( (delta * l_rate) / (SAMPLES * BATCH) )  - sparsity; 
	//b[h_idx] += ( (delta * l_rate) / (SAMPLES * BATCH) ); 
} 

/* --------------------------------------------------------------- 
*    TRANSPOSE 
* Coalesced transpose with no bank conflicts. 
* 
* w  | float* | Weights 
* wt | float* | Weights Transposed 
------------------------------------------------------------------*/ 
__global__ void transpose(float *w, float *wt) 
{ 
	__shared__ float tile[TILE_DIM][TILE_DIM+1]; 
	int xIndex = blockIdx.x * TILE_DIM + threadIdx.x; 
	int yIndex = blockIdx.y * TILE_DIM + threadIdx.y; 
	int index_in = xIndex + (yIndex)*HSIZE; 

	xIndex = blockIdx.y * TILE_DIM + threadIdx.x; 
	yIndex = blockIdx.x * TILE_DIM + threadIdx.y; 
	int index_out = xIndex + (yIndex)*VSIZE; 

	for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) 
	{ 
		tile[threadIdx.y+i][threadIdx.x] = w[index_in+i*HSIZE]; 
	} 

	__syncthreads(); 

	for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) 
	{ 
		wt[index_out+i*VSIZE] = tile[threadIdx.x][threadIdx.y+i]; 
	} 

	//CUPRINTF("\tw[0] is:%f\n", w[0]);
	//CUPRINTF("\twt[0] is:%f\n", wt[0]);
} 

